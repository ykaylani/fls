#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

__global__ void normalize(float* __restrict__ leq, float* __restrict__ distF, int tcells) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= tcells) return;

    for(int i = 0; i < latSize; i++) {

        distF[i * tcells + idx] += (leq[idx * latSize + i] - distF[idx * latSize + i]) / rtime;
    }
}

__global__ void stream(float* distF, int tcells) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= tcells) return;


}