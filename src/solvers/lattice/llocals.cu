#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../lattice/llutcs.h"

__device__ void leqstates(float* __restrict__ feq, float* __restrict__ ldensity, float3* __restrict__ lvelocity, int tcells) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= tcells) return;

    #pragma unroll
    for (int i = 0; i < latSize; i++) {
        int fi =  i * tcells + idx;

        float eiudot = fmaf(lutDirections[i].x, lvelocity[idx].x, 0.0);
        eiudot = fmaf(lutDirections[i].y, lvelocity[idx].y, eiudot);
        eiudot = fmaf(lutDirections[i].z, lvelocity[idx].z, eiudot);

        float udot = fmaf(lvelocity[idx].x, lvelocity[idx].x, 0.0);
        udot = fmaf(lvelocity[idx].y, lvelocity[idx].y, udot);
        udot = fmaf(lvelocity[idx].z, lvelocity[idx].z, udot);

        feq[fi] = lutWeights[i] * ldensity[idx] * (1 + 3 * eiudot + (9.0f / 2.0f) * (eiudot * eiudot) - (3.0f / 2.0f) * udot); //c^2 is 0.3... in D3Q19
    }
}

__global__ void lstates(float* __restrict__ distF, float* __restrict__ feq, float* __restrict__ ldensity, float3* __restrict__ lvelocity, int tcells, bool init) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= tcells) return;

    #pragma unroll
    for (int i = 0; i < latSize; i++) {
        ldensity[idx] += distF[i * tcells + idx];

        float3 r = float3(lutDirections[i].x * distF[i * tcells + idx], lutDirections[i].y * distF[i * tcells + idx], lutDirections[i].z * distF[ i * tcells + idx]);
        lvelocity[idx].x += r.x;
        lvelocity[idx].y += r.y;
        lvelocity[idx].z += r.z;
    }

    float invdensity = 1.0f / fmaxf(ldensity[idx], 1e-20f);
    lvelocity[idx].x *= invdensity;
    lvelocity[idx].y *= invdensity;
    lvelocity[idx].z *= invdensity;

    if (init) { leqstates(feq, ldensity, lvelocity, tcells); } //init does not cause warp divergence; it is a switch for leqstates (same value for all threads)
}