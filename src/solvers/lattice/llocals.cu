#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../lattice/lutcs.h"

__device__ void leqstates(float* __restrict__ feq, float* __restrict__ ldensity, float3* __restrict__ lvelocity, int tcells) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= tcells) return;

    for (int i = 0; i < latSize; i++) {
        int fi =  i * tcells + idx;

        float eiudot = lutDirections[i].x * lvelocity[idx].x + lutDirections[i].y * lvelocity[idx].y + lutDirections[i].z * lvelocity[idx].z;
        float udot = lvelocity[idx].x * lvelocity[idx].x + lvelocity[idx].y * lvelocity[idx].y + lvelocity[idx].z * lvelocity[idx].z;
        feq[fi] = lutWeights[i] * ldensity[idx] * (1 + 3 * eiudot + (9.0f / 2.0f) * (eiudot * eiudot) - (3.0f / 2.0f) * udot); //s is 0.3... in D3Q19
    }
}

__global__ void lstates(float* __restrict__ distF, float* __restrict__ feq, float* __restrict__ ldensity, float3* __restrict__ lvelocity, int tcells) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= tcells) return;

    for (int i = 0; i < latSize; i++) {
        ldensity[idx] += distF[i * tcells + idx];

        float3 r = float3(lutDirections[i].x * distF[i * tcells + idx], lutDirections[i].y * distF[i * tcells + idx], lutDirections[i].z * distF[ i * tcells + idx]);
        lvelocity[idx].x += r.x;
        lvelocity[idx].y += r.y;
        lvelocity[idx].z += r.z;
    }

    float invdensity = 1.0f / fmaxf(ldensity[idx], 1e-20f);
    lvelocity[idx].x *= invdensity;
    lvelocity[idx].y *= invdensity;
    lvelocity[idx].z *= invdensity;

    leqstates(feq, ldensity, lvelocity, tcells);
}