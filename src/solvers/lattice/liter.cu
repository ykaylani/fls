#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../../../include/internal/llutcs.h"

__global__ void collide(float* __restrict__ leq, float* __restrict__ distF, int tcells) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= tcells) return;

    #pragma unroll
    for(int i = 0; i < latSize; i++) {

        distF[i * tcells + idx] += (leq[i * tcells + idx] - distF[i * tcells + idx]) / rtime;
    }
}

__global__ void stream(float* distF, int tcells, float* out) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= tcells) return;

    #pragma unroll
    for (int i = 0; i < latSize; i++) {
        out[i * tcells + idx] = distF[i * tcells + (idx + lutOffsets[i])];
    }
}