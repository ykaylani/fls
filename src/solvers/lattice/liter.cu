#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

__global__ void normalize(float* __restrict__ leq, float* __restrict__ distF, int tcells) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= tcells) return;

    #pragma unroll
    for(int i = 0; i < latSize; i++) {

        distF[i * tcells + idx] += (leq[i * tcells + idx] - distF[i * tcells + idx]) / rtime;
    }
}

__global__ void stream(float* distF, int tcells) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= tcells) return;


}