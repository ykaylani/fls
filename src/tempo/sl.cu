#include "hip/hip_runtime.h"
#include "../../include/internal/sdatal.h"
#include "../solvers/lattice/llocals.cu"
#include "../solvers/lattice/iter.cu"

const int3 lattice = int3(128,128,128);

int tcells = lattice.x * lattice.y * lattice.z;
int blocksize = 512;
int gridSize = (tcells + blocksize - 1) / blocksize;

void klbm() {

    sdatal data; data.init(lattice);
    lstates<<<gridSize, blocksize>>>(data.distF, data.leq, data.ldensity, data.lvelocity, tcells, true);
    normalize<<<gridSize, blocksize>>>(data.leq, data.distF, tcells);
    data.destroy();
}

int main() {

    klbm();
    return 0;
}

